/*
    tetmag - A general-purpose finite-element micromagnetic simulation software package
    Copyright (C) 2016-2023 CNRS and Université de Strasbourg

    Author: Riccardo Hertel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as
    published by the Free Software Foundation, either version 3 of the
    License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details. 
 
    Contact: Riccardo Hertel, IPCMS Strasbourg, 23 rue du Loess, 
    	     67034 Strasbourg, France.
	     riccardo.hertel@ipcms.unistra.fr
	     
    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/
 
/*
 * SpMatCUDA.cu
 *
 *  Created on: Sep 17, 2020
 *      Author: hertel
 */

#include "SpMatCUDA.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime_api.h>

using namespace Eigen;
// typedef SparseMatrix<double, ColMajor> SpMat_CM;
typedef thrust::device_vector<double> devVecD;

// construct thrust vector from Eigen Vector:
devVecD devVecXd(const Eigen::VectorXd& eig) {
   devVecD dv(std::vector<double>(eig.data(), eig.data() + eig.size()));
   return dv;
}


template<class deviceType>
void SpMatCUDA::delete_vec( deviceType& x_d ) {
	x_d.clear();
	x_d.shrink_to_fit();
	x_d.~device_vector();
}


SpMatCUDA::SpMatCUDA() : alpha(1.0), beta(0.0) {}


SpMatCUDA::SpMatCUDA( const SpMat& m1_) : alpha(1.0), beta(0.0) {
	SpMat_CM m1 = m1_; // copy into / enforce ColMajor format
	m1.makeCompressed();
	nnz = m1.nonZeros();
	cols = m1.cols();
	rows = m1.rows();
	Eigen::VectorXd cscValA_h    = Map<VectorXd>( m1.valuePtr(), nnz );
	Eigen::VectorXi	cscRowIndA_h = Map<VectorXi>( m1.innerIndexPtr(), nnz );
	Eigen::VectorXi cscColPtrA_h = Map<VectorXi>( m1.outerIndexPtr(), m1.outerSize() + 1 );
	cscVals_d.resize(cscValA_h.size());
	cscVals_d = devVecD ( cscValA_h.data(), cscValA_h.data() + cscValA_h.size() );
	cscCols_d.resize(cscColPtrA_h.size());
	cscCols_d = devVecI ( cscColPtrA_h.data(), cscColPtrA_h.data() + cscColPtrA_h.size() );
	cscRows_d.resize(cscRowIndA_h.size());
	cscRows_d = devVecI ( cscRowIndA_h.data(), cscRowIndA_h.data() + cscRowIndA_h.size() );
	setOnDev();
}


void SpMatCUDA::checkStatusCusparse(hipsparseStatus_t& status) {
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("CUSPARSE API failed at line %d with error: %s (%d)\n",
	   __LINE__, hipsparseGetErrorString(status), status);
  } //else { std::cout << "CuSparseStatus = 0K" << std::endl;  }
}


void SpMatCUDA::setOnDev() {
  handle = NULL;
  hipsparseCreate(&handle);
  hipsparseStatus_t status;
  dBuffer = NULL;
  bufferSize = 0;
  VectorXd zero = VectorXd::Zero(rows);

  //	hipsparseStatus_t status __attribute__ ((unused)) ; // this was in the old version.

  // prepare sparse matrix:
  status =  hipsparseCreateCsr(&matA, rows, cols, nnz,
			      thrust::raw_pointer_cast(cscCols_d.data()),
			      thrust::raw_pointer_cast(cscRows_d.data()),
			      thrust::raw_pointer_cast(cscVals_d.data()),
			      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
			      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  assert(status == HIPSPARSE_STATUS_SUCCESS);
  checkStatusCusparse(status);

  // prepare output vector:
  hipMalloc((void**) &dY, rows * sizeof(double));
  hipMemcpy(dY, zero.data(), rows * sizeof(double), hipMemcpyHostToDevice);
  hipsparseCreateDnVec(&vecY, rows, dY, HIP_R_64F);

  // prepare input vector:
  zero.resize(cols);
  hipMalloc((void**) &dX, cols * sizeof(double));
  hipMemcpy(dX, zero.data(), cols * sizeof(double), hipMemcpyHostToDevice);
  hipsparseCreateDnVec(&vecX, cols, dX, HIP_R_64F);

  // prepare buffer:
  status = hipsparseSpMV_bufferSize( handle, HIPSPARSE_OPERATION_TRANSPOSE,
				   &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
#if (CUDART_VERSION > 11000)
				   HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) ;
#else
                                   HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) ;
#endif
  
  checkStatusCusparse(status);
  hipMalloc(&dBuffer, bufferSize);
}


void SpMatCUDA::mvp() {
  hipsparseStatus_t stat = hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
				       &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
#if (CUDART_VERSION > 11000)
				   HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) ;
#else
                                   HIPSPARSE_MV_ALG_DEFAULT, dBuffer) ;
#endif
//  checkStatusCusparse(stat);
}


devVecD SpMatCUDA::mvp(const devVecD& x) {
	setX(x);
	mvp();
	return mvpResDev();
}


void SpMatCUDA::setX(const VectorXd& x) {
  hipMemcpy(dX, x.data(), x.size() * sizeof(double),  hipMemcpyHostToDevice);
}


void SpMatCUDA::setX(const devVecD& x) {
  hipMemcpy(dX, thrust::raw_pointer_cast(x.data()), x.size() * sizeof(double),  hipMemcpyDeviceToDevice);
}


VectorXd  SpMatCUDA::mvpResEig(){
  VectorXd output(rows);
   hipMemcpy(output.data(), dY, rows * sizeof(double),  hipMemcpyDeviceToHost);
   return output;
}


devVecD SpMatCUDA::mvpResDev(){
	return thrust::device_vector< double > (dY, dY + rows);
}


SpMatCUDA::~SpMatCUDA() {
	hipsparseDestroySpMat (matA);
	hipsparseDestroyDnVec (vecX);
	hipsparseDestroyDnVec (vecY);
	hipsparseDestroy (handle);
	delete_vec(cscVals_d);
	delete_vec(cscCols_d);
	delete_vec(cscRows_d);
}

