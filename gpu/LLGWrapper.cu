#include "hip/hip_runtime.h"
/*
    tetmag - A general-purpose finite-element micromagnetic simulation software package
    Copyright (C) 2016-2023 CNRS and Université de Strasbourg

    Author: Riccardo Hertel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as
    published by the Free Software Foundation, either version 3 of the
    License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details. 
 
    Contact: Riccardo Hertel, IPCMS Strasbourg, 23 rue du Loess, 
    	     67034 Strasbourg, France.
	     riccardo.hertel@ipcms.unistra.fr
	     
    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

/*
 * LLGWrapper.cu
 *
 *  Created on: Sep 23, 2019
 *      Author: riccardo
 */

// Cannot be included in TheLLG class: Separate translation unit needed for nvcc.

#include "LLGWrapper.h"
#include <functional>
#include <thrust/copy.h>
#include <nvector/nvector_cuda.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

dev_vec mag_vec_tmp;
dev_vec ret_vec_tmp;

// using namespace boost::numeric::odeint;
//typedef runge_kutta_fehlberg78<dev_vec, double, dev_vec, double> fehlberg78_gpu;
//typedef controlled_runge_kutta< fehlberg78_gpu > fehlberg78_controlled_gpu;


void LLGWrapper::init(const int nx) {
	nx3 = 3 * nx;
	mag_vec_tmp.resize(nx3);
	ret_vec_tmp.resize(nx3);
}


dev_vec TheLLG::sttDynamics_GPU(const dev_vec &mag_vec) {
	dev_vec LLGpart_d = classicVersion_GPU(mag_vec);
	stt.Ustt = gpucalc->UTermSTT_GPU();
	dev_vec ret_vec_d = *gpucalc->STT_term_LLG_dev(stt.Ustt, alpha, stt.beta);
	if (stt.pulseIsUsed) {
		double pulseVal = stt.gaussPulseValue(timeInPs);
		thrust::transform(ret_vec_d.begin(), ret_vec_d.end(), ret_vec_d.begin(), pulseVal * thrust::placeholders::_1);
	}
	thrust::transform(thrust::device, ret_vec_d.begin(), ret_vec_d.end(), LLGpart_d.begin(), ret_vec_d.begin(), thrust::plus<double>());
	return(ret_vec_d);
}

Eigen::MatrixXd TheLLG::effFieldsForGPU(const dev_vec &mag_vec) {
// This should be improved. There are too many copy operations.	
// An overlaod of evaluateAllEffectiveFields() could be prepared
// that would take a device vector as argument.

	std::vector<double> mag_vec_h(3 * nx);
	thrust::copy(mag_vec.begin(), mag_vec.end(), mag_vec_h.begin());
	Eigen::Map<const MatrixXd_CM> Mag(mag_vec_h.data(), nx, 3);
	evaluateAllEffectiveFields(Mag);
	return totalEffectiveField();
// One should further check if it is necessary to return MatrixXd 
// in the individual field calculations. 
// It should be possible to leave the data on the device.
}

dev_vec TheLLG::classicVersion_GPU(const dev_vec &mag_vec) {
	gpucalc->setMagDev(mag_vec);
	Heff = effFieldsForGPU(mag_vec);
	return *gpucalc->ClassicLLG_dev(Heff, alpha);
}


dev_vec TheLLG::noPrecession_GPU(const dev_vec &mag_vec) {
	gpucalc->setMagDev(mag_vec);
	Heff = effFieldsForGPU(mag_vec);
	return *gpucalc->LLG_noPrec_dev(Heff, alpha);
}


template<class deviceType>
void delete_vec( deviceType& x_d ) {
	x_d.clear();
	x_d.shrink_to_fit();
	x_d.~device_vector();
}

LLGWrapper::~LLGWrapper(){
	hipDeviceSynchronize();
	delete_vec(mag_vec_tmp);
	delete_vec(ret_vec_tmp);
}

void TheLLG::operator()(const thrust::device_vector<double>& mag_d, thrust::device_vector<double>& dxdt_d, const double theTime /*time*/) {
	timeInPs = theTime / realTimeScale;
	dxdt_d = selectedLLGType_GPU(mag_d);
}


void TheLLG::selectLLGTypeGPU(int choice) {
	enum choices { noPrec , usualLLG , STT };
	useSTT = false;
		if (choice == noPrec) {
			selectedLLGType_GPU = [this](const dev_vec &m) -> dev_vec { return noPrecession_GPU(m); };
		} else if (choice == STT ) {
			useSTT = true;
			selectedLLGType_GPU = [this](const dev_vec &m) -> dev_vec { return sttDynamics_GPU(m); };
		} else	{
			selectedLLGType_GPU = [this](const dev_vec &m) -> dev_vec {	return classicVersion_GPU(m); };
		}
}


int TheLLG::IntegrateOnGPU(std::vector<double>& mag_vec, double ode_start_t, double ode_end_t, double dt ) {
// Not referenced any more. Used to be called from TheSimulation::start() but was removed due to THRUST / ODEINT incompatibility
	std::reference_wrapper<TheLLG>  LLGRef = std::ref(*this);
	copyTimer.start();
	mag_vec_tmp = mag_vec; // copy to device vector
    copyTimer.add();
//    int its = boost::numeric::odeint::integrate<double>(LLGRef, mag_vec_tmp, ode_start_t, ode_start_t + ode_end_t, dt );
    copyTimer.start();
    thrust::copy(mag_vec_tmp.begin(), mag_vec_tmp.end(), mag_vec.begin());
    copyTimer.add();
//    return its;
	return 0;
}


int TheLLG::gpuODE(std::vector<double>& mag_vec, double ode_start_t, double ode_end_t, double dt ) {
	(void) dt;
	long int its_l, its_nl = 0;
	std::shared_ptr<UserData> data = alloc_user_data(nx);
	int flag;
	sunindextype N = 3 * nx;
    copyTimer.start();
//
//	int* major = (int*)malloc(sizeof(int));
//	int* minor = (int*)malloc(sizeof(int)); 
//	int* patch = (int*)malloc(sizeof(int));
//	int major ;
//	int minor ;
//	int patch ;
//	int len;
//	char label;
//	char* label = (char *)malloc( 20 *sizeof(char));;
//	SUNDIALSGetVersionNumber(major, minor, patch, label, len);
//	int SUNDIALSGetVersionNumber(major, minor, 	patch, 	&label, len);
//	std::cout << "CVODE version : " << major << "." << minor << "." << patch << std::endl;
//	std::exit(1);
//	


	mag_vec_tmp = mag_vec;  // copy to thrust::device_vector
#ifndef OLD_CVODE_VERSION
	SUNContext sunctx; 
    SUNContext_Create(NULL, &sunctx); 
	N_Vector m_gpu = N_VMake_Cuda(N, mag_vec.data(), thrust::raw_pointer_cast(mag_vec_tmp.data()), sunctx); 
#else	
    N_Vector m_gpu = N_VMake_Cuda(N, mag_vec.data(), thrust::raw_pointer_cast(mag_vec_tmp.data())); 
#endif	
    copyTimer.add();

// copy to N_V
//	hipMemcpy(mdata, thrust::raw_pointer_cast(mag3_d.data()), N * sizeof(realtype), hipMemcpyDeviceToDevice);
	void *cvode_mem = NULL;	
	#ifndef OLD_CVODE_VERSION
		cvode_mem = CVodeCreate(CV_ADAMS, sunctx); 
	#else
		cvode_mem = CVodeCreate(CV_ADAMS); 
	#endif
//	cvode_mem = CVodeCreate(CV_BDF);
	flag = CVodeInit(cvode_mem, rhs_d, ode_start_t, m_gpu);
	
	// CV_BDF configuration:
	//	realtype abstol = 1.e-8;
	//	realtype reltol = 1.e-5;

	// CV_ADAMS configuration:
	realtype abstol = 1.e-6;
	realtype reltol = 1.e-6;
	flag = CVodeSStolerances(cvode_mem, reltol, abstol);
	flag = CVodeSetUserData(cvode_mem, data.get());

	SUNLinearSolver LS;
	#if OLD_CVODE_VERSION
		LS = SUNLinSol_SPGMR(m_gpu, PREC_NONE, 0); 	
	#else		
		LS = SUNLinSol_SPGMR(m_gpu, PREC_NONE, 0, sunctx); 	
	#endif
	flag = CVodeSetLinearSolver(cvode_mem, LS, NULL);

	realtype tout = ode_end_t + ode_start_t;
	realtype t = ode_start_t;

	flag = CVode(cvode_mem, tout, m_gpu, &t, CV_NORMAL);
	if (flag) {std::cerr << "Warning: GPU integration failed." << std::endl; }
	CVodeGetNumNonlinSolvIters(cvode_mem, &its_nl);
	CVodeGetNumLinIters(cvode_mem, &its_l);
	int its = static_cast<int>(its_nl + its_l); // iterations
	double* res_p = N_VGetDeviceArrayPointer_Cuda(m_gpu);
	copyTimer.start();
	thrust::copy(res_p, res_p + N, mag_vec_tmp.begin()); // copy from N_V
	thrust::copy(mag_vec_tmp.begin(), mag_vec_tmp.end(), mag_vec.begin()); // copy to std::vector -- returned by ref.
	copyTimer.add();
//  The magnetization needs to be transfered to EffFieldGPU in order to calculate max.torque
	gpucalc->setMagDev(mag_vec_tmp);
//	N_VCopyFromDevice_Cuda(m);
//	double * res_p = N_VGetHostArrayPointer_Cuda(m);
//	thrust::copy(res_p, res_p + N, mag_vec.begin());
	N_VDestroy_Cuda(m_gpu);
	CVodeFree(&cvode_mem);
	SUNLinSolFree(LS);
	#ifndef OLD_CVODE_VERSION
		SUNContext_Free(&sunctx); 
	#endif 
	return its;
}


int TheLLG::rhs_d(realtype t, N_Vector u, N_Vector u_dot, void *user_data) {
	(void) t;
	UserData *u_data;
	u_data = (UserData*) user_data;
	sunindextype N = 3 * u_data->nx;
	realtype *dudata = N_VGetDeviceArrayPointer_Cuda(u_dot);
	realtype *udata =  N_VGetDeviceArrayPointer_Cuda(u);
	u_data->llg->copyTimer.start();
	thrust::copy(udata, udata + N, mag_vec_tmp.begin());
	u_data->llg->copyTimer.add();
	u_data->llg->operator()(mag_vec_tmp, ret_vec_tmp, t); // integrate
	u_data->llg->copyTimer.start();
	hipMemcpy(dudata, thrust::raw_pointer_cast(ret_vec_tmp.data()), N * sizeof(realtype), hipMemcpyDeviceToDevice);
	u_data->llg->copyTimer.add();
	return (0); // <--- This is required to signal success
}
