#include "hip/hip_runtime.h"
/*
    tetmag - A general-purpose finite-element micromagnetic simulation software package
    Copyright (C) 2016-2023 CNRS and Université de Strasbourg

    Author: Riccardo Hertel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as
    published by the Free Software Foundation, either version 3 of the
    License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details. 
 
    Contact: Riccardo Hertel, IPCMS Strasbourg, 23 rue du Loess, 
    	     67034 Strasbourg, France.
	     riccardo.hertel@ipcms.unistra.fr
	     
    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

/*
 * EffFieldGPU.cu
 *
 *  Created on: Sep 4, 2019
 *      Author: riccardo
 */

#include "typedefs.h"
#include "EffFieldGPU.h"
#include <hipsparse.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
using namespace Eigen;


enum Coords {
	x, y, z
};


EffFieldGPU::EffFieldGPU(const int nx_) : nx(nx_) {
	copyTimer.reset();
	init(nx);
}

void EffFieldGPU::setUniaxialAnisotropy(const MatrixXd& KuAxis_eig, const VectorXd& Ku1) {
	kuAxis_x = std::make_shared<dev_vec>(nx);
	kuAxis_y = std::make_shared<dev_vec>(nx);
	kuAxis_z = std::make_shared<dev_vec>(nx);
	kuVal = std::make_shared<dev_vec>(nx);
	thrust::copy(KuAxis_eig.col(x).data(), KuAxis_eig.col(x).data() + nx, kuAxis_x->begin());
	thrust::copy(KuAxis_eig.col(y).data(), KuAxis_eig.col(y).data() + nx, kuAxis_y->begin());
	thrust::copy(KuAxis_eig.col(z).data(), KuAxis_eig.col(z).data() + nx, kuAxis_z->begin());
	thrust::copy(Ku1.data(), Ku1.data() + nx , kuVal->begin());
}

void EffFieldGPU::setCubicAnisotropy(const std::vector<Matrix3d>& cubicAxes, const VectorXd& Kc1, const VectorXd& Kc2) {
// to be completed
}


Matrix<value_type, Dynamic, Dynamic> EffFieldGPU::ExchangeFieldGPU() {

	*Hxcx_d = xc_cuda->mvp(*mx_d);
	*Hxcy_d = xc_cuda->mvp(*my_d);
	*Hxcz_d = xc_cuda->mvp(*mz_d);

	copyTimer.start();
	thrust::copy( Hxcx_d->begin(), Hxcx_d->end() , Hxc_unrolled.data() );
  	thrust::copy( Hxcy_d->begin(), Hxcy_d->end() , Hxc_unrolled.data() + nx );
	thrust::copy( Hxcz_d->begin(), Hxcz_d->end() , Hxc_unrolled.data() + 2 * nx );
	copyTimer.end();
	copyTimer.add();

	return Map<Matrix<value_type, Dynamic, Dynamic> >( Hxc_unrolled.data(), nx, 3 );

}


template<class T>
struct sumOfThree {
    __host__ __device__
        T operator()(const thrust::tuple<T,T,T>& a) const
        {
            return thrust::get<0>(a)  +    // first vector
                   thrust::get<1>(a)  +    // second vector
                   thrust::get<2>(a) ;     // third vector
        }
};

dev_vec EffFieldGPU::cwiseProduct(const dev_vec& v1, const dev_vec& v2) {
  dev_vec out(v1.size());
  thrust::transform( v1.begin(), v1.end(), v2.begin(), out.begin(), thrust::multiplies<value_type>() );
  return out;
}


Matrix<double, Dynamic, Dynamic> EffFieldGPU::UTermSTT_GPU() {

  *u_xx = GradX_cuda->mvp(*mx_d);
  *u_xy = GradY_cuda->mvp(*mx_d);
  *u_xz = GradZ_cuda->mvp(*mx_d);

  *u_yx = GradX_cuda->mvp(*my_d);
  *u_yy = GradY_cuda->mvp(*my_d);
  *u_yz = GradZ_cuda->mvp(*my_d);

  *u_zx = GradX_cuda->mvp(*mz_d);
  *u_zy = GradY_cuda->mvp(*mz_d);
  *u_zz = GradZ_cuda->mvp(*mz_d);
  
  *ju_xx = cwiseProduct( *eta_jx_d, *u_xx );
  *ju_xy = cwiseProduct( *eta_jy_d, *u_xy );
  *ju_xz = cwiseProduct( *eta_jz_d, *u_xz );

  *ju_yx = cwiseProduct( *eta_jx_d, *u_yx );
  *ju_yy = cwiseProduct( *eta_jy_d, *u_yy );
  *ju_yz = cwiseProduct( *eta_jz_d, *u_yz );
  
  *ju_zx = cwiseProduct( *eta_jx_d, *u_zx );
  *ju_zy = cwiseProduct( *eta_jy_d, *u_zy );
  *ju_zz = cwiseProduct( *eta_jz_d, *u_zz );
 
  thrust::transform( thrust::make_zip_iterator(thrust::make_tuple(ju_xx->begin(), ju_xy->begin(), ju_xz->begin())),
                       thrust::make_zip_iterator(thrust::make_tuple(ju_xx->end(), ju_xy->end(), ju_xz->end())),
                       u_term_stt->begin(),
                       sumOfThree<value_type>() );
  thrust::transform( thrust::make_zip_iterator(thrust::make_tuple(ju_yx->begin(), ju_yy->begin(), ju_yz->begin())),
                       thrust::make_zip_iterator(thrust::make_tuple(ju_yx->end(), ju_yy->end(), ju_yz->end())),
					   u_term_stt->begin() + nx,
                       sumOfThree<value_type>() );
  thrust::transform( thrust::make_zip_iterator(thrust::make_tuple(ju_zx->begin(), ju_zy->begin(), ju_zz->begin())),
                       thrust::make_zip_iterator(thrust::make_tuple(ju_zx->end(), ju_zy->end(), ju_zz->end())),
					   u_term_stt->begin() + 2 * nx,
                       sumOfThree<value_type>() );

  thrust::copy(u_term_stt->begin(), u_term_stt->end(), retMatXd.data());
  return retMatXd;
}

void EffFieldGPU::calcCurlM() {

  *cmx1 = tGradY_cuda->mvp(*mz_d);
  *cmx2 = tGradZ_cuda->mvp(*my_d);
  *cmy1 = tGradZ_cuda->mvp(*mx_d);
  *cmy2 = tGradX_cuda->mvp(*mz_d);
  *cmz1 = tGradX_cuda->mvp(*my_d);
  *cmz2 = tGradY_cuda->mvp(*mx_d);

  thrust::transform( cmx1->begin(), cmx1->end(), cmx2->begin(), curlM->begin(), thrust::minus<value_type>() );
  thrust::transform( cmy1->begin(), cmy1->end(), cmy2->begin(), curlM->begin() + nx, thrust::minus<value_type>() );
  thrust::transform( cmz1->begin(), cmz1->end(), cmz2->begin(), curlM->begin() + 2 * nx, thrust::minus<value_type>() );
}

void EffFieldGPU::setMagDev( MRef& Mag ) {
	copyTimer.start();
	thrust::copy(Mag.col(x).data(), Mag.col(x).data() + nx, mx_d->begin());
	thrust::copy(Mag.col(y).data(), Mag.col(y).data() + nx, my_d->begin());
	thrust::copy(Mag.col(z).data(), Mag.col(z).data() + nx, mz_d->begin());

	thrust::copy( mx_d->begin(), mx_d->end(), mag3_n->begin() );
	thrust::copy( my_d->begin(), my_d->end(), mag3_n->begin() + nx );
	thrust::copy( mz_d->begin(), mz_d->end(), mag3_n->begin() + 2 * nx );
	copyTimer.end();
   copyTimer.add();
}

void EffFieldGPU::setMagDev( const std::vector<value_type>& mag_vec ) {
  nx = mag_vec.size() / 3;
  copyTimer.start();
  thrust::copy( mag_vec.begin(), mag_vec.begin() + nx , mx_d->begin() );
  thrust::copy( mag_vec.begin() + nx, mag_vec.begin() + 2 * nx, my_d->begin() );
  thrust::copy( mag_vec.begin() + 2 * nx, mag_vec.end() , mz_d->begin() );

  thrust::copy( mx_d->begin(), mx_d->end(), mag3_n->begin() );
  thrust::copy( my_d->begin(), my_d->end(), mag3_n->begin() + nx );
  thrust::copy( mz_d->begin(), mz_d->end(), mag3_n->begin() + 2 * nx );
  copyTimer.end();
  copyTimer.add();
}

void EffFieldGPU::setMagDev( const devVecD& mag_vec ) {
  nx = mag_vec.size() / 3;
  copyTimer.start();
  thrust::copy( mag_vec.begin(), mag_vec.begin() + nx , mx_d->begin() );
  thrust::copy( mag_vec.begin() + nx, mag_vec.begin() + 2 * nx, my_d->begin() );
  thrust::copy( mag_vec.begin() + 2 * nx, mag_vec.end() , mz_d->begin() );

  thrust::copy( mx_d->begin(), mx_d->end(), mag3_n->begin() );
  thrust::copy( my_d->begin(), my_d->end(), mag3_n->begin() + nx );
  thrust::copy( mz_d->begin(), mz_d->end(), mag3_n->begin() + 2 * nx );
  copyTimer.end();
  copyTimer.add();
}

void EffFieldGPU::setGradientMatsOnDev( const SpMat& tGradX, const SpMat& tGradY, const SpMat& tGradZ ) {
	tGradX_cuda = std::make_shared<SpMatCUDA>( tGradX );
	tGradY_cuda = std::make_shared<SpMatCUDA>( tGradY );
	tGradZ_cuda = std::make_shared<SpMatCUDA>( tGradZ );
}

void EffFieldGPU::setExchangeMatOnDev(const SpMat& XC_h) {
	xc_cuda = std::make_shared<SpMatCUDA>( XC_h );
	assert (nx == XC_h.rows()) ;
}

void EffFieldGPU::init(int nx) {
	Hxc_unrolled.resize(3*nx);

	mx_d = std::make_shared<dev_vec>(nx);
	my_d = std::make_shared<dev_vec>(nx);
	mz_d = std::make_shared<dev_vec>(nx);
	heffx_d = std::make_shared<dev_vec>(nx);
	heffy_d = std::make_shared<dev_vec>(nx);
	heffz_d = std::make_shared<dev_vec>(nx);
	dxdt = std::make_shared<dev_vec>(3 * nx);
	mag3_n = std::make_shared<dev_vec>(3 * nx);

	dmi3 = std::make_shared<dev_vec>(3 * nx);
	cmx1 = std::make_shared<dev_vec>(nx);
	cmx2 = std::make_shared<dev_vec>(nx);
	cmy1 = std::make_shared<dev_vec>(nx);
	cmy2 = std::make_shared<dev_vec>(nx);
	cmz1 = std::make_shared<dev_vec>(nx);
	cmz2 = std::make_shared<dev_vec>(nx);
	curlM = std::make_shared<dev_vec>(3 * nx);
	surfTerm = std::make_shared<dev_vec>(3 * nx);
	Hxcx_d = std::make_shared<dev_vec>(nx);
	Hxcy_d = std::make_shared<dev_vec>(nx);
	Hxcz_d = std::make_shared<dev_vec>(nx);
	hani_x = std::make_shared<dev_vec>(nx);
	hani_y = std::make_shared<dev_vec>(nx);
	hani_z = std::make_shared<dev_vec>(nx);
	kx_mx = std::make_shared<dev_vec>(nx);
	ky_my = std::make_shared<dev_vec>(nx);
	kz_mz = std::make_shared<dev_vec>(nx);
	tmp0 = std::make_shared<dev_vec>(nx);
	tmp1 = std::make_shared<dev_vec>(nx);
	retVecLLG.resize(3*nx);
//	retVecLLG_d.resize(3*nx);
	retMatXd.resize(nx,3);
}

void EffFieldGPU::setSTTDataOnDevice(const SpMat& GradX, const SpMat& GradY, const SpMat& GradZ,
				     const VectorXd& eta_jx, const VectorXd& eta_jy, const VectorXd& eta_jz) {

	GradX_cuda = std::make_shared<SpMatCUDA>( GradX );
	GradY_cuda = std::make_shared<SpMatCUDA>( GradY );
	GradZ_cuda = std::make_shared<SpMatCUDA>( GradZ );
	ustt_d = std::make_shared<dev_vec>( 3 * nx );
	eta_jx_d = std::make_shared<dev_vec>(nx);
	eta_jy_d = std::make_shared<dev_vec>(nx);
	eta_jz_d = std::make_shared<dev_vec>(nx);
	thrust::copy( eta_jx.data(), eta_jx.data() + nx, eta_jx_d->begin() );
	thrust::copy( eta_jy.data(), eta_jy.data() + nx, eta_jy_d->begin() );
	thrust::copy( eta_jz.data(), eta_jz.data() + nx, eta_jz_d->begin() );
	u_xx = std::make_shared<dev_vec>(nx);
	u_xy = std::make_shared<dev_vec>(nx);
	u_xz = std::make_shared<dev_vec>(nx);
	u_yx = std::make_shared<dev_vec>(nx);
	u_yy = std::make_shared<dev_vec>(nx);
	u_yz = std::make_shared<dev_vec>(nx);
	u_zx = std::make_shared<dev_vec>(nx);
	u_zy = std::make_shared<dev_vec>(nx);
	u_zz = std::make_shared<dev_vec>(nx);

	ju_xx = std::make_shared < dev_vec >(nx);
	ju_xy = std::make_shared < dev_vec >(nx);
	ju_xz = std::make_shared < dev_vec >(nx);
	ju_yx = std::make_shared < dev_vec >(nx);
	ju_yy = std::make_shared < dev_vec >(nx);
	ju_yz = std::make_shared < dev_vec >(nx);
	ju_zx = std::make_shared < dev_vec >(nx);
	ju_zy = std::make_shared < dev_vec >(nx);
	ju_zz = std::make_shared < dev_vec >(nx);

	u_term_stt = std::make_shared < dev_vec >( 3 * nx );
}

void EffFieldGPU::setDMIdata(const VectorXd& D, const VectorXd& invNodeVol,const MatrixXd& nv_n, const VectorXd& nodesurface_h) {
	assert(nx == D.size());
	dev_vec nv_x(nx), nv_y(nx), nv_z(nx);
	dev_vec dmival(nx), invnodevol(nx);
	dev_vec nodesurface(nx);
	thrust::copy(nodesurface_h.data(), nodesurface_h.data() + nx, nodesurface.begin());
	thrust::copy(nv_n.col(x).data(), nv_n.col(x).data() + nx, nv_x.begin());
	thrust::copy(nv_n.col(y).data(), nv_n.col(y).data() + nx, nv_y.begin());
	thrust::copy(nv_n.col(z).data(), nv_n.col(z).data() + nx, nv_z.begin());
	nv_surf_x = std::make_shared<dev_vec>(nx);
	nv_surf_y = std::make_shared<dev_vec>(nx);
	nv_surf_z = std::make_shared<dev_vec>(nx);
	*nv_surf_x = cwiseProduct(nv_x, nodesurface);
	*nv_surf_y = cwiseProduct(nv_y, nodesurface);
	*nv_surf_z = cwiseProduct(nv_z, nodesurface);

	thrust::copy(invNodeVol.data(), invNodeVol.data() + nx, invnodevol.begin());
	thrust::copy(D.data(), D.data() + nx, dmival.begin());
	dev_vec dmi_factor = cwiseProduct(invnodevol, dmival);
	dev_vec dmi_fac3 = dmi_factor;
	dmi_fac3.reserve(2 * nx);
	dmi_fac3.insert(dmi_fac3.end(), dmi_factor.begin(), dmi_factor.end());
	dmi_fac3.insert(dmi_fac3.end(), dmi_factor.begin(), dmi_factor.end()); // yes, do it twice.
	dmi_fac = std::make_shared<dev_vec>(dmi_fac3.size());
	*dmi_fac = dmi_fac3;
};

struct LLG_functor {
  const value_type alpha;
  LLG_functor( const value_type alpha_ )  : alpha(alpha_) {}
    template < class Tuple >
    __device__    
    void operator() ( Tuple t ) const  {

      Matrix< value_type , 3 , 1> M, Heff, MxH, dMdt;
      M(x) = thrust::get<0>( t );
      M(y) = thrust::get<1>( t );
      M(z) = thrust::get<2>( t );

      Heff(x) = thrust::get<6>(t);
      Heff(y) = thrust::get<7>(t);
      Heff(z) = thrust::get<8>(t);

      MxH = M.cross(Heff);

      dMdt = -MxH - alpha * M.cross(MxH);
      thrust::get<3>(t) = dMdt(x);
      thrust::get<4>(t) = dMdt(y);
      thrust::get<5>(t) = dMdt(z);
    }
};

struct LLGnoPrec_functor {
  const value_type alpha;
  LLGnoPrec_functor( const value_type alpha_ )  : alpha(alpha_) {}
    template < class Tuple >
    __device__    
    void operator() ( Tuple t ) const  {
      Matrix< value_type , 3 , 1> M, Heff, MxH, dMdt;

      M(x) = thrust::get<0>( t );
      M(y) = thrust::get<1>( t );
      M(z) = thrust::get<2>( t );

      Heff(x) = thrust::get<6>( t );
      Heff(y) = thrust::get<7>( t );
      Heff(z) = thrust::get<8>( t );

      dMdt = - alpha * M.cross(M.cross(Heff));
      thrust::get<3>(t) = dMdt(x);
      thrust::get<4>(t) = dMdt(y);
      thrust::get<5>(t) = dMdt(z);
    }
};

std::vector<value_type> EffFieldGPU::LLG_noPrec_hst(MRef &Heff,	const value_type alpha) {
	dxdt = LLG_noPrec_dev(Heff, alpha);
	copyTimer.start();
	thrust::copy(dxdt->begin(), dxdt->end(), retVecLLG.begin());
	copyTimer.add();
	return retVecLLG;
}

std::shared_ptr<dev_vec> EffFieldGPU::LLG_noPrec_dev(MRef &Heff, const value_type alpha) {
  thrust::copy( Heff.col(x).data(), Heff.col(x).data() + nx, heffx_d->begin() );
  thrust::copy( Heff.col(y).data(), Heff.col(y).data() + nx, heffy_d->begin() );
  thrust::copy( Heff.col(z).data(), Heff.col(z).data() + nx, heffz_d->begin() );

  thrust::for_each(
		  thrust::make_zip_iterator(
				  thrust::make_tuple(
						  mx_d->begin(),  my_d->begin(),  mz_d->begin() ,
						  dxdt->begin(),
						  dxdt->begin() + nx,
						  dxdt->begin() + 2 * nx,
						  heffx_d->begin(), heffy_d->begin(), heffz_d->begin() )) ,
		  thrust::make_zip_iterator(
				  thrust::make_tuple(
						  mx_d->end(),  my_d->end(), mz_d->end(),
						  dxdt->begin() + nx ,
						  dxdt->begin() + 2 * nx,
						  dxdt->begin() + 3 * nx,
						  heffx_d->end(),  heffy_d->end(), heffz_d->end() )),
				  LLGnoPrec_functor(alpha) );
   return dxdt;
}



std::vector<value_type> EffFieldGPU::ClassicLLG_hst(MRef& Heff, const value_type alpha) {
	dxdt = ClassicLLG_dev(Heff, alpha);
	copyTimer.start();
	thrust::copy(dxdt->begin(), dxdt->end(), retVecLLG.begin());
	copyTimer.add();
	return retVecLLG;
}


std::shared_ptr<dev_vec> EffFieldGPU::ClassicLLG_dev(MRef& Heff, const value_type alpha) {
	copyTimer.start();
	thrust::copy(Heff.col(x).data(), Heff.col(x).data() + nx, heffx_d->begin());
	thrust::copy(Heff.col(y).data(), Heff.col(y).data() + nx, heffy_d->begin());
	thrust::copy(Heff.col(z).data(), Heff.col(z).data() + nx, heffz_d->begin());
	copyTimer.end();
	copyTimer.add();

	thrust::for_each(
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->begin(), my_d->begin(), mz_d->begin(),
							dxdt->begin(),
							dxdt->begin() + nx,
							dxdt->begin() + 2 * nx,
							heffx_d->begin(), heffy_d->begin(), heffz_d->begin())),
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->end(), my_d->end(), mz_d->end(),
							dxdt->begin() + nx,
							dxdt->begin() + 2 * nx,
							dxdt->begin() + 3 * nx, heffx_d->end(),
							heffy_d->end(), heffz_d->end())),
			LLG_functor(alpha));
	return dxdt;
}



struct STT_functor {
  const value_type alpha;
  const value_type beta;
  STT_functor( const value_type alpha_, const value_type beta_ )  : alpha(alpha_), beta(beta_) {}
    template < class Tuple >
    __device__    
    void operator() ( Tuple t ) const  {

      Matrix< value_type , 3 , 1> M, Ustt, MxU, dMdt;
      M(x) = thrust::get<0>( t );
      M(y) = thrust::get<1>( t );
      M(z) = thrust::get<2>( t );

      Ustt(x) = thrust::get<6>( t );
      Ustt(y) = thrust::get<7>( t );
      Ustt(z) = thrust::get<8>( t );

      MxU = M.cross(Ustt);

      dMdt = -(beta - alpha) * MxU - (1 + alpha * beta) * M.cross(MxU);
      thrust::get<3>( t ) = dMdt(x);
      thrust::get<4>( t ) = dMdt(y);
      thrust::get<5>( t ) = dMdt(z);
    }
};


std::shared_ptr<dev_vec> EffFieldGPU::STT_term_LLG_dev(MRef &Ustt, const value_type alpha, const value_type beta) {

	thrust::copy(Ustt.col(x).data(), Ustt.col(z).data() + nx, ustt_d->begin());
	thrust::for_each(
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->begin(), my_d->begin(), mz_d->begin(),
							dxdt->begin(), dxdt->begin() + nx, dxdt->begin() + 2 * nx,
							ustt_d->begin(), ustt_d->begin() + nx, ustt_d->begin() + 2 * nx)),
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->end(), my_d->end(), mz_d->end(),
							dxdt->begin() + nx, dxdt->begin() + 2 * nx,	dxdt->begin() + 3 * nx,
							ustt_d->begin() + nx, ustt_d->begin() + 2 * nx, ustt_d->begin() + 3 * nx)),
			STT_functor(alpha, beta));
	return dxdt;
}


std::vector<value_type> EffFieldGPU::STT_term_LLG_hst(MRef &Ustt, const value_type alpha, const value_type beta) {
	dxdt = STT_term_LLG_dev(Ustt, alpha, beta);
	thrust::copy(dxdt->begin(), dxdt->end(), retVecLLG.begin());
	return retVecLLG;
}

struct torque {
  __host__ __device__
  value_type operator() (const thrust::tuple<value_type, value_type, value_type, value_type, value_type, value_type> T ) const  {

    Matrix<value_type , 3 , 1> M, H;
    thrust::tie(M(x), M(y), M(z), H(x), H(y), H(z)) = T;
    return (M.cross(H)).norm();
  }
};

value_type EffFieldGPU::MaxTorque(MRef& Heff) {

	copyTimer.start();
	thrust::copy( Heff.col(x).data(), Heff.col(x).data() + nx, heffx_d->begin() );
	thrust::copy( Heff.col(y).data(), Heff.col(y).data() + nx, heffy_d->begin() );
	thrust::copy( Heff.col(z).data(), Heff.col(z).data() + nx, heffz_d->begin() );
	copyTimer.end();
	copyTimer.add();
	value_type mt = thrust::transform_reduce(
				thrust::make_zip_iterator( thrust::make_tuple(
						mx_d->begin(), my_d->begin(), mz_d->begin(),
						heffx_d->begin(), heffy_d->begin(), heffz_d->begin())),
				thrust::make_zip_iterator( thrust::make_tuple(
						mx_d->end(), my_d->end(), mz_d->end(),
						heffx_d->end(), heffy_d->end(), heffz_d->end())),
						torque(),
						0.0, thrust::maximum<value_type>());
	return mt;
}

Matrix<double, Dynamic,Dynamic> EffFieldGPU::CubicAnisotropyField()	{
	MatrixXd retHcub = MatrixXd::Zero(nx, 3);
	// to be completed

	dev_vec dea_x(nx), dea_y(nx), dea_z(nx);
	dev_vec alf_x(nx), alf_y(nx), alf_z(nx);
	dev_vec a0(nx), a1(nx), a2(nx);
/*
    Vector3d dea;
	for (int i = 0; i < nx; ++i) {
		Vector3d alf = cubicAxes[i].transpose() * Mag.row(i).transpose();
		for (int j = 0; j < 3; ++j) {
			double a0 = alf(j % 3);
			double a1 = alf((j + 1) % 3);
			double a2 = alf((j + 2) % 3);
		    dea(j) = -2. * a0 * (Kc1(i) * (a1 * a1 + a2 * a2) + Kc2(i) * a1 * a1 * a2 * a2);
		}
		Hcub.row(i) = cubicAxes[i] * dea;
	}
*/
	return retHcub;
}


Matrix<double, Dynamic, Dynamic> EffFieldGPU::UniaxialAnisotropyField() {

	*kx_mx = cwiseProduct( *kuAxis_x, *mx_d );
	*ky_my = cwiseProduct( *kuAxis_y, *my_d );
	*kz_mz = cwiseProduct( *kuAxis_z, *mz_d );

	thrust::transform(
			thrust::make_zip_iterator(thrust::make_tuple(kx_mx->begin(), ky_my->begin(), kz_mz->begin())),
	                       thrust::make_zip_iterator(thrust::make_tuple(kx_mx->end(), ky_my->end(), kz_mz->end())),
	                       tmp0->begin(),
	                       sumOfThree<value_type>() );
	*tmp1 = cwiseProduct(*kuVal,*tmp0);
	thrust::transform(tmp1->begin(), tmp1->end(), tmp1->begin(), 2. * thrust::placeholders::_1);

	*hani_x = cwiseProduct( *kuAxis_x, *tmp1 );
	*hani_y = cwiseProduct( *kuAxis_y, *tmp1 );
	*hani_z = cwiseProduct( *kuAxis_z, *tmp1 );

	thrust::copy( hani_x->begin(), hani_x->end(), retMatXd.col(x).data() );
	thrust::copy( hani_y->begin(), hani_y->end(), retMatXd.col(y).data() );
	thrust::copy( hani_z->begin(), hani_z->end(), retMatXd.col(z).data() );
	return retMatXd;
}


struct Cross_GPU {
    template < class Tuple >
    __device__
    void operator() ( Tuple t ) const  {
      Matrix<value_type , 3 , 1> a, b, axb;

      a(x) = thrust::get<0>( t );
      a(y) = thrust::get<1>( t );
      a(z) = thrust::get<2>( t );

      b(x) = thrust::get<3>( t );
      b(y) = thrust::get<4>( t );
      b(z) = thrust::get<5>( t );

      axb = a.cross(b);
      thrust::get<6>( t ) = axb(x);
      thrust::get<7>( t ) = axb(y);
      thrust::get<8>( t ) = axb(z);
    }
};


struct DMI_OP {
	template<class Tuple>
	__host__ __device__
	value_type operator()(Tuple t) const {
		value_type v1, v2, v3;
		thrust::tie(v1, v2, v3) = t;
		return (v1 - 2. * v2) * v3;
	}
};


Matrix<double, Dynamic, Dynamic> EffFieldGPU::DMIField() {
//	Hdmi = (surfIntDMI(Mag) - 2. * CurlM(Mag)).array().colwise() * invNodeVol.cwiseProduct(D).array(); //

	calcCurlM();

	thrust::for_each(
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->begin(), my_d->begin(), mz_d->begin(),
							nv_surf_x->begin(), nv_surf_y->begin(), nv_surf_z->begin(),
							surfTerm->begin(), surfTerm->begin() + nx, surfTerm->begin() + 2 * nx)),
			thrust::make_zip_iterator(
					thrust::make_tuple(mx_d->end(), my_d->end(), mz_d->end(),
							nv_surf_x->end(), nv_surf_y->end(), nv_surf_z->end(),
							surfTerm->begin() + nx, surfTerm->begin() + 2 * nx, surfTerm->begin() + 3 * nx) ),
							Cross_GPU());

	thrust::transform(
			thrust::make_zip_iterator(
					thrust::make_tuple(surfTerm->begin(), curlM->begin(),dmi_fac->begin())),
			thrust::make_zip_iterator(
					thrust::make_tuple(surfTerm->end(), curlM->end(), dmi_fac->end())),
					dmi3->begin(),
					DMI_OP());

	thrust::host_vector<value_type> ret_h = *dmi3;
	return Map<Matrix<value_type, Dynamic, Dynamic> >(ret_h.data(), nx, 3);
}


struct normalize {
	template<class Tuple>
	__host__ __device__
	void operator()( Tuple t) {
		Matrix<value_type , 3 , 1> M;
		thrust::tie( M(x), M(y), M(z) ) = t;
		M.normalize();
		thrust::get<x>(t) = M(x);
		thrust::get<y>(t) = M(y);
		thrust::get<z>(t) = M(z);
	}
};



void EffFieldGPU::NormalizeMag( MatrixXd& Mag, int nx ) {
	copyTimer.start();
	thrust::copy(Mag.col(x).data(), Mag.col(x).data() + nx, mx_d->begin());
	thrust::copy(Mag.col(y).data(), Mag.col(y).data() + nx, my_d->begin());
	thrust::copy(Mag.col(z).data(), Mag.col(z).data() + nx, mz_d->begin());
	copyTimer.end();
	copyTimer.add();

	thrust::for_each(
				thrust::make_zip_iterator(
						thrust::make_tuple(mx_d->begin(), my_d->begin(), mz_d->begin())),
				thrust::make_zip_iterator(
						thrust::make_tuple(mx_d->end(), my_d->end(), mz_d->end())),
				normalize());
	copyTimer.start();
	thrust::copy(mx_d->begin(), mx_d->end(), Mag.col(x).data());
	thrust::copy(my_d->begin(), my_d->end(), Mag.col(y).data());
	thrust::copy(mz_d->begin(), mz_d->end(), Mag.col(z).data());
	thrust::copy(mx_d->begin(), mx_d->end(), mag3_n->begin());
	thrust::copy(my_d->begin(), my_d->end(), mag3_n->begin() + nx);
	thrust::copy(mz_d->begin(), mz_d->end(), mag3_n->begin() + 2 * nx);
	copyTimer.end();
	copyTimer.add();
}

void EffFieldGPU::displayTimer() {
	  std::cout << "GPU time for copying data [s]:\t" << copyTimer.durationInMus() / 1.e6 << std::endl;
}

