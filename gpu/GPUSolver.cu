#include "hip/hip_runtime.h"
/*
    tetmag - A general-purpose finite-element micromagnetic simulation software package
    Copyright (C) 2016-2023 CNRS and Université de Strasbourg

    Author: Riccardo Hertel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as
    published by the Free Software Foundation, either version 3 of the
    License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details. 
 
    Contact: Riccardo Hertel, IPCMS Strasbourg, 23 rue du Loess, 
    	     67034 Strasbourg, France.
	     riccardo.hertel@ipcms.unistra.fr
	     
    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

/*
 * GPUSolver.cu
 *
 *  Created on: Sep 2, 2020
 *      Author: hertel
 */

#include "GPUSolver.h"
#include <Eigen/Dense>
#include <Eigen/SparseCore>
#include <iostream>
#include <cmath>
GPUSolver::GPUSolver() {}


GPUSolver::~GPUSolver() {}


void GPUSolver::init() {
    hipsparseCreate(&bprm.cusparse_handle);
    prm.precond.coarsening.relax = 0.75;
    prm.solver.tol  = SolverFactory::cgTol;
    prm.solver.maxiter = 100;
//    std::cout << "CG solver tolerance: " << std::scientific << prm.solver.tol  << std::fixed << std::endl;
}


void GPUSolver::report() {
  std::cout << *solver << std::endl;
}


bool GPUSolver::setup(SpMat& A_) {
	bool success = true;
    SpMat_RM A = A_; // copy into / enforce RowMajor format
    A.makeCompressed();
    int nnz = A.nonZeros();
    std::vector<int> col(nnz);
    std::vector<int> ptr(A.outerSize() + 1);
    std::vector<double> val(nnz);
    n = A.cols();
    x.resize(n);
    x_d.resize(n);
    b_d.resize(n);

    std::copy( A.valuePtr(), A.valuePtr( ) + nnz , val.begin());
    std::copy( A.innerIndexPtr(), A.innerIndexPtr() + nnz, col.begin() );
    std::copy( A.outerIndexPtr(), A.outerIndexPtr() + A.outerSize() + 1 , ptr.begin() );
    try {
    solver = std::make_shared<Solver>(std::tie(n,  ptr, col,  val), prm, bprm);
    } catch (...) {
    	success = false;
    }
    return success;
}


void GPUSolver::solve() {
	thrust::copy(b.data(), b.data() + n, b_d.begin());
	std::tie(iters, error) = solver->operator()(b_d, x_d);
	thrust::copy(x_d.begin(), x_d.end(), x.data());
}


bool GPUSolver::compute() {
	init();
	return setup(A);
}


bool GPUSolver::wasSuccessful(){
	return std::isfinite(error);
}

