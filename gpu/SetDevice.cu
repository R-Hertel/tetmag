/*
    tetmag - A general-purpose finite-element micromagnetic simulation software package
    Copyright (C) 2016-2023 CNRS and Université de Strasbourg

    Author: Riccardo Hertel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as
    published by the Free Software Foundation, either version 3 of the
    License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details. 
 
    Contact: Riccardo Hertel, IPCMS Strasbourg, 23 rue du Loess, 
    	     67034 Strasbourg, France.
	     riccardo.hertel@ipcms.unistra.fr
	     
    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

/*
 * SetDevice.cu
 *
 *  Created on: Sep 16, 2020
 *      Author: hertel
 */

#include "SetDevice.h"
#include <iostream>

SetDevice::SetDevice(int mydev_) : mydev(mydev_){
	success = false;
	ndev = 0;
}


bool SetDevice::set(){
	success = true;
  hipError_t err = hipGetDeviceCount(&ndev);

  if (err != hipSuccess) {
    std::cout << "No GPU device available. Defaulting to CPU computation" << std::endl;
    success = false;
    return success;
  }
// else {
//    std::cout << "Found " << ndev << " devices." << std::endl;
//  }

  if (mydev > ndev - 1 | mydev < 0) {
    std::cout << "device number " << mydev << " not available. Defaulting to device number 0." << std::endl;
    mydev = 0;
  }
  hipSetDevice(mydev);
  return success;
}
